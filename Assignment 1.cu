/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n). 
 * Note: All lines marked in --> should be replaced with code. 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>   // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....

    //we are assuming to calculate the values by assuming the A matrix as blocks and B matrix as threads per block
    // so we are calclating the blockid and threadid for coorecsponding block here in ROW wise manner and multiplying
    // correctly to get the C value and findingthe C index by that id and putting in right place

    long int id = blockIdx.x;
    long int id2 = threadIdx.x;
    for(long int t1=0;t1<n;t1++){
        for(long int t2=0;t2<n;t2++){
            long int c_index = id*m*n*n+t2*m*n+id2+t1*m;
            long int a_index = id*n+t1;
            long int b_index = id2*n+t2;
            C[c_index] = A[a_index]*B[b_index];
        }
    }
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....

//we are assuming to calculate the values by assuming the A matrix as blocks and B matrix as threads per block
    // so we are calclating the blockid and threadid for coorecsponding block here in COLUMN wise manner and multiplying
    // correctly to get the C value and findingthe C index by that id and putting in right place

    long int id = blockIdx.x;
    long int id2 = threadIdx.x;
    for(long int t1=0;t1<m;t1++){
        for(long int t2=0;t2<m;t2++){
            long int c_index = id*m+t2+id2*m*n+t1*m*n*n;
            long int a_index = id+t1*n;
            long int b_index = id2+t2*n;
            C[c_index] = A[a_index]*B[b_index];
        }
    }
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
    // long int id = blockIdx.x*blockDim.x+threadIdx.x;
    long long id1 = blockIdx.x*gridDim.y+blockIdx.y;
    long int id2 = id1*blockDim.x*blockDim.y;
    long int id3 = id2+threadIdx.x*blockDim.y+threadIdx.y;

    if(id3 < m*n*m*n){
        // printf("%3d\n",id3);

       long int row = id3/(m*n);
       long int col = id3%(m*n);

        long int a_row = row/n;
        long int a_col = col/m;
        long int b_row = col%m;
        long int b_col = row%n;

        long int a_index = a_row*n+a_col;
        long int b_index = b_row*n+b_col;

        C[id3] = A[a_index]*B[b_index];

    }
}

/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(long int *arr, long int rows, long int cols, char* filename){
    outfile.open(filename);
    for(long int i = 0; i < rows; i++){
        for(long int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    long int m,n;	
    cin>>m>>n;	

    // Host_arrays 
    long int *h_a,*h_b,*h_c;

    // Device arrays 
    long int *d_a,*d_b,*d_c;
	
    // Allocating space for the host_arrays 
    h_a = (long int *) malloc(m * n * sizeof(long int));
    h_b = (long int *) malloc(m * n * sizeof(long int));	
    h_c = (long int *) malloc(m * m * n * n * sizeof(long int));	

    // Allocating memory for the device arrays 

    // --> Allocate memory for A on device 
    hipMalloc(&d_a,m*n*sizeof(long int));
    // --> Allocate memory for B on device 
    hipMalloc(&d_b,m*n*sizeof(long int));
    // --> Allocate memory for C on device 
    hipMalloc(&d_c,m*m*n*n*sizeof(long int));

    // Read the input matrix A 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_a[i];
    }

    //Read the input matrix B 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_b[i];
    }

    // Transfer the input host arrays to the device 
    // --> Copy A from Host to Device
    hipMemcpy(d_a,h_a,m*n*sizeof(long int),hipMemcpyHostToDevice);
    // --> Copy B from Host to Device 
    hipMemcpy(d_b,h_b,m*n*sizeof(long int),hipMemcpyHostToDevice);

    long int gridDimx, gridDimy;
    
    // Launch the kernels
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    // --> Set the launch configuration 

    double starttime = rtclock();  

    // --> Launch the kernel 
    per_row_AB_kernel<<<m,m>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize();                                                           

    double endtime = rtclock(); 
	  printtime("GPU Kernel-1 time: ", starttime, endtime);  

    // --> Copy C from Device to Host 
    hipMemcpy(h_c,d_c,m*m*n*n*sizeof(long int),hipMemcpyDeviceToHost);
    

    printMatrix(h_c, m * n, m * n,"kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/
    
    // --> Set the launch configuration
    dim3 block2(n,2,1);

    starttime = rtclock(); 

    // --> Launch the kernel 
    per_column_AB_kernel<<<n,block2>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize(); 

    endtime = rtclock(); 
  	printtime("GPU Kernel-2 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    hipMemcpy(h_c,d_c,m*m*n*n*sizeof(long int),hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output 
     **/
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid3(gridDimx,gridDimy,1);
    dim3 block3(64,16,1);

    starttime = rtclock();  

    // --> Launch the kernel 
    per_element_kernel<<<grid3,block3>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize();                                                              

    endtime = rtclock();  
	printtime("GPU Kernel-3 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    hipMemcpy(h_c,d_c,m*m*n*n*sizeof(long int),hipMemcpyDeviceToHost);
    printMatrix(h_c, m * n, m * n,"kernel3.txt");

    return 0;
}

